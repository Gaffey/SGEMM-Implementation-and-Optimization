#include "hip/hip_runtime.h"
#include "utils.cpp"

const int TILE_SIZE = 16;

template <typename T>
__global__ void matmul_naive(T* a, T* b, T* c, int M, int K, int N) {
    /* A naive implementation of matrix multiplication.
     * a: MxK
     * b: KxN
     * c: MxN
     */
    // If the whole threads can't cover the matrix elements,
    // the outside loop is required.
    // Here I only consider the case that the size of the matrix
    // is multiple of block size.
    int j = threadIdx.x + blockIdx.x*blockDim.x;
    int i = threadIdx.y + blockIdx.y*blockDim.y;

	// A for loop in one thread caculates the 
	// one value in output matrix.
	T elem = 0;
	for (int k = 0; k < K; ++k) {
		elem = elem + a[i*M+k]*b[k*K+j];
	}
	c[i*M+j] = elem;
}

int main(int argc, char *argv[]) {
    int M = std::atoi(argv[1]), K = std::atoi(argv[2]), N = std::atoi(argv[3]);
	dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 blocksPerGrid(N / TILE_SIZE, M / TILE_SIZE);

    double* a = utils::random_matrix_gpu<double>(M, K, utils::C_ORDER);
    double* b = utils::random_matrix_gpu<double>(K, N, utils::C_ORDER);
    double* c = new double[M*N];
    double *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, M*K*sizeof(double));
    hipMalloc((void**)&dev_b, K*N*sizeof(double));
    hipMalloc((void**)&dev_c, M*N*sizeof(double));

    hipMemcpy(dev_a, a, M*K*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, K*N*sizeof(double), hipMemcpyHostToDevice);
    matmul_naive<double><<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c, M, K, N);
    hipMemcpy(c, dev_c, M*N*sizeof(double), hipMemcpyDeviceToHost);

#ifdef CHECK
    std::cout << (utils::check_mul<double>(a, b, c, M, K, N, utils::C_ORDER)
		    ? "Correct!!" : "Wrong Answer!") << std::endl;
#endif
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    free(a);
    free(b);
    free(c);
    return 0;
}





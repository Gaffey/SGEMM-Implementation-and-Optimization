#include "hip/hip_runtime.h"
#include "../utils.cpp"
#include <stdio.h>

__global__ void add (int *a, int *b, int *c) {
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
	c[tid] = a[tid]+b[tid];
}

bool check_sum(int *c, int *c_, int n) {
    for (int i = 0; i < n; ++i) {
	if (c[i] != c_[i])
	    return false;
    }
    return true;
}

int main(int argc, char *argv[]) {
    const int N = std::atoi(argv[1]);
    const int TILE_SIZE = 16;
    int a[N], b[N], c[N], c_[N];
    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, N*sizeof(int));
    hipMalloc((void**)&dev_b, N*sizeof(int));
    hipMalloc((void**)&dev_c, N*sizeof(int));

    for (int i = 0; i < N; ++i) {
        a[i] = i;
        b[i] = i;
	    c_[i] = a[i] + b[i];
    }
    
    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

    add<<<N / TILE_SIZE, TILE_SIZE>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
	printf("Naive addition:");
    printf(check_sum(c, c_, N) ? "Correct!\n" : "Wrong...\n");

#ifdef DEBUG
    for (int i = 0; i < N; ++i) {
        printf("%d + %d = %d\n", a[i], b[i], c_op[i]); 
    }std::cout << std::endl;
#endif

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}


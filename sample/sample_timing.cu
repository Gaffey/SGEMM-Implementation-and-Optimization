#include "utils.cpp"
#include "blas.cu"
#include <stdio.h>

#define N 10

/* CUDA offers a relatively light-weight alternative to
     * CPU timers via the CUDA event API
     * The logic of usage is as follows.
     */

int main() {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;
    
    // Create CUDA event obeject by hipEvent_t
    // Use hipEventCreate API to create    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc((void**)&dev_a, N*sizeof(int));
    hipMalloc((void**)&dev_b, N*sizeof(int));
    hipMalloc((void**)&dev_c, N*sizeof(int));

    for (int i = 0; i < N; ++i) {
        a[i] = i;
        b[i] = i*i;
    }
    
    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

    // Squeeze the function to time by invoke API
    // hipEventRecord twice.
    // Kernel call is asynchronized to host code.
    hipEventRecord(start);
    add<<<N, 1>>>(dev_a, dev_b, dev_c, N);
    hipEventRecord(stop);

    // Need to block CPU execution until 'stop' is record, which
    // means the execution on GPU is completed.
    hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    for (int i = 0; i < N; ++i) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]); 
    }std::cout << std::endl;
   
    // Refer a float type variable to calculate
    // the elapsed time. 
    float t = 0;
    hipEventElapsedTime(&t, start, stop);
    printf("add takes %f ms to complete\n", t);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}


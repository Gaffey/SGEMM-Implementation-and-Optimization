#include "utils.cpp"
#include "hipblas.h"
#include <hip/hip_runtime.h>

int main(int argc, char *argv[]) {
    int M = std::atoi(argv[1]), K = std::atoi(argv[2]), N = std::atoi(argv[3]);
    double *a = utils::random_matrix_gpu<double>(M, K, utils::FORTRAN_ORDER);
    double *b = utils::random_matrix_gpu<double>(K, N, utils::FORTRAN_ORDER);
    double *c = new double[M*N];

    double *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, M*K*sizeof(double));
    hipMalloc((void**)&dev_b, K*N*sizeof(double));
    hipMalloc((void**)&dev_c, M*N*sizeof(double));

    hipMemcpy(dev_a, a, M*K*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, K*N*sizeof(double), hipMemcpyHostToDevice);
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    double al=1.0f, bet=0;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, 
		    &al, dev_a, M, dev_b, K, &bet, dev_c, M);

    hipMemcpy(c, dev_c, M*N*sizeof(double), hipMemcpyDeviceToHost);
#ifdef CHECK
    std::cout << (utils::check_mul<double>(a, b, c, M, K, N, utils::FORTRAN_ORDER) 
		    ? "Correct!!" : "Wrong Answer!") << std::endl;
#endif
#ifdef DEBUG
    std::cout << "Matrix A:" << std::endl;
    utils::print_mat_gpu(a, M, K, utils::FORTRAN_ORDER);
    std::cout << "\nMatrix B:" << std::endl;
    utils::print_mat_gpu(b, K, N, utils::FORTRAN_ORDER);
    std::cout << "\nMatrix C:" << std::endl;
    utils::print_mat_gpu(c, M, N, utils::FORTRAN_ORDER);
#endif

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    free(a);
    free(b);
    free(c);
    return 0;
}
    

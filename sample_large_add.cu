#include "hip/hip_runtime.h"
#include "utils.cpp"
#include "blas.cu"
#include <stdio.h>

#define N 100000

__global__ void large_scale_add(int *a, int *b, int *c) {
    /* For 100,000 elements, this functions takes about 0.01ms
     * to complete.
     */
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += gridDim.x*blockDim.x;
    }
}

int main() {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc((void**)&dev_a, N*sizeof(int));
    hipMalloc((void**)&dev_b, N*sizeof(int));
    hipMalloc((void**)&dev_c, N*sizeof(int));

    for (int i = 0; i < N; ++i) {
        a[i] = i;
        b[i] = i;
    }
    
    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

   
    hipEventRecord(start);
    large_scale_add<<<256, 256>>>(dev_a, dev_b, dev_c);
    hipEventRecord(stop);
 
    hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    /* 
    for (int i = 0; i < N; ++i) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]); 
    }std::cout << std::endl;*/
   
    float t = 0;
    hipEventElapsedTime(&t, start, stop);
    printf("add takes %f ms to complete\n", t);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}


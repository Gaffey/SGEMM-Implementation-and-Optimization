#include "utils.cpp"
#include "blas.cu"
#include <stdio.h>

#define N 10

int main() {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, N*sizeof(int));
    hipMalloc((void**)&dev_b, N*sizeof(int));
    hipMalloc((void**)&dev_c, N*sizeof(int));

    for (int i = 0; i < N; ++i) {
        a[i] = i;
        b[i] = i*i;
    }
    
    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

    add<<<N, 1>>>(dev_a, dev_b, dev_c, N);

    hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]); 
    }std::cout << std::endl;

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}


#include "hip/hip_runtime.h"
#include "utils.cpp"

const int TILE_SIZE = 16;

template <typename T>
__global__ void matmul_Tiling(T *A, T *B, T *C, int M, int K, int N) {
	/* Basic tiling implementation of matrix multiplication.
	 * Based on a more mathematically reasonable indexing method.
	 */
	int bx = blockIdx.x, by = blockIdx.y;
	int tx = threadIdx.x, ty = threadIdx.y;

	__shared__ T As[TILE_SIZE][TILE_SIZE];
	__shared__ T Bs[TILE_SIZE][TILE_SIZE];

	int aBegin = K * TILE_SIZE * by;
	int aEnd = aBegin + K - 1;
	int aStep = TILE_SIZE;

	int bBegin = TILE_SIZE * bx;
	int bStep = TILE_SIZE * N;

	T Csub = 0;

	for (int i = aBegin, j = bBegin; i <= aEnd; i += aStep, j += bStep) {
		As[ty][tx] = A[i + K * ty + tx];
		Bs[tx][ty] = B[j + N * tx + ty];

		__syncthreads();

		for (int k = 0; k < TILE_SIZE; ++k) {
			Csub += As[ty][k]*Bs[k][tx];
		}
		
		__syncthreads();
	}
	int cIdx = N * TILE_SIZE * by + TILE_SIZE * bx;
	C[cIdx + N * ty + tx] = Csub;
}

int main(int argc, char *argv[]) {
	int M = std::atoi(argv[1]);
	int K = std::atoi(argv[2]);
	int N = std::atoi(argv[3]);

	dim3 threads(TILE_SIZE, TILE_SIZE);
	dim3 grid(N / TILE_SIZE, M / TILE_SIZE);

	double *a = utils::random_matrix_gpu<double>(M, K, utils::C_ORDER);
	double *b = utils::random_matrix_gpu<double>(K, N, utils::C_ORDER);
	double *c = new double[M*N];
	
	double *dev_a, *dev_b, *dev_c;

	hipMalloc((void**)&dev_a, M*K*sizeof(double));
	hipMalloc((void**)&dev_b, K*N*sizeof(double));
	hipMalloc((void**)&dev_c, M*N*sizeof(double));

	hipMemcpy(dev_a, a, M*K*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, K*N*sizeof(double), hipMemcpyHostToDevice);

	matmul_Tiling<double><<<grid, threads>>>(dev_a, dev_b, dev_c, M, K, N);

	hipMemcpy(c, dev_c, M*N*sizeof(double), hipMemcpyDeviceToHost);

	std::cout << (utils::check_mul<double>(a, b, c, M, K, N, utils::C_ORDER) ? "Correct!!" : "Wrong Answer!") << std::endl;

#ifdef DEBUG
    std::cout << "Matrix A:" << std::endl;
    utils::print_mat_gpu(a, M, K, utils::C_ORDER);
    std::cout << "Matrix B:" << std::endl;
    utils::print_mat_gpu(b, K, N, utils::C_ORDER);
    std::cout << "Matrix C:" << std::endl;
    utils::print_mat_gpu(c, M, N, utils::C_ORDER);
#endif
	return 0;
}


#include "hip/hip_runtime.h"
#include "utils.cpp"

dim3 threadsPerBlock(16, 16);

template <typename T>
__global__ void matmul_naive(T* a, T* b, T* c, int M, int K, int N) {
    /* A naive implementation of matrix multiplication.
     * a: MxK
     * b: KxN
     * c: MxN
     * 
     * Average Time: 1000x1000x1000, 4.85s
     * Average Time: 1024x1024x1024, 1.53s 
     */
    // If the whole threads can't cover the matrix elements,
    // the outside loop is required.
    // Here I only consider the case that the size of the matrix
    // is multiple of block size.
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;

    for (int i = x; i < M; i += blockDim.x) {
	for (int j = y; j < N; j += blockDim.y) {
	    c[i*M+j] = 0;
	    // A for loop in one thread caculates the 
	    // one value in output matrix.
	    for (int k = 0; k < K; ++k) {
		c[i*M+j] += a[i*M+k]*b[k*K+j];
	    }
	}
    }
}

int main(int argc, char *argv[]) {
    int M = std::atoi(argv[1]), K = std::atoi(argv[2]), N = std::atoi(argv[3]);
    dim3 blocksPerGrid;
    blocksPerGrid.x = M / threadsPerBlock.x;
    blocksPerGrid.y = N / threadsPerBlock.y;
    blocksPerGrid.z = 1;

    double* a = utils::random_matrix_gpu<double>(M, K, utils::C_ORDER);
    double* b = utils::random_matrix_gpu<double>(K, N, utils::C_ORDER);
    double* c = new double[M*N];
    double *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, M*K*sizeof(double));
    hipMalloc((void**)&dev_b, K*N*sizeof(double));
    hipMalloc((void**)&dev_c, M*N*sizeof(double));

    hipMemcpy(dev_a, a, M*K*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, K*N*sizeof(double), hipMemcpyHostToDevice);
    matmul_naive<double><<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c, M, K, N);
    hipMemcpy(c, dev_c, M*N*sizeof(double), hipMemcpyDeviceToHost);

    std::cout << (utils::check_mul<double>(a, b, c, M, K, N, utils::C_ORDER)
		    ? "Correct!!" : "Wrong Answer!") << std::endl;

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    free(a);
    free(b);
    free(c);
    return 0;
}




